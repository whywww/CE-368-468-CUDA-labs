#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	const int TILE_WIDTH = 32;
	int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;

	float Pvalue = 0.0;

	__shared__ float Ms[TILE_WIDTH*TILE_WIDTH], Ns[TILE_WIDTH*TILE_WIDTH];
	
	for (int i = 0; i < M.width/TILE_WIDTH + 1; i++){
		// Zero the paddings
		Ms[threadIdx.y*TILE_WIDTH + threadIdx.x] = 0.0;
		Ns[threadIdx.y*TILE_WIDTH + threadIdx.x] = 0.0;

		// Copy from global to share memory
		if (Row < M.height && i*TILE_WIDTH + threadIdx.x < M.width){
			Ms[threadIdx.y*TILE_WIDTH + threadIdx.x] = M.elements[Row*M.width + i*TILE_WIDTH + threadIdx.x];
		} 

		if (Col < N.width && i*TILE_WIDTH + threadIdx.y < N.height){
			Ns[threadIdx.y*TILE_WIDTH + threadIdx.x] = N.elements[(i*TILE_WIDTH + threadIdx.y)*N.width + Col];
		}
		__syncthreads();

		// Compute on shared memory
		for (int k = 0; k < TILE_WIDTH; k++){
			float M_elem = Ms[threadIdx.y*TILE_WIDTH + k];
			float N_elem = Ns[k*TILE_WIDTH + threadIdx.x];
			Pvalue += M_elem * N_elem;
		}
		__syncthreads();
	}
	// Copy back to global memory
	if (Row < P.height && Col < P.width){
		P.elements[Row*P.width + Col] = Pvalue;
	}
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
