#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	//const int TILE_WIDTH = 128 / sizeof(float);
	const int TILE_WIDTH = 32;
	int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;

	float Pvalue = 0;

	// Copy from global to share memory
	__shared__ float Ms[TILE_WIDTH*TILE_WIDTH], Ns[TILE_WIDTH*TILE_WIDTH], Ps[TILE_WIDTH*TILE_WIDTH];
	Ms[Row*TILE_WIDTH+Col] = M.elements[Row*TILE_WIDTH+Col];
	Ns[Row*TILE_WIDTH+Col] = N.elements[Row*TILE_WIDTH+Col];
	//Ps[Row*TILE_WIDTH+Col] = P.elements[Row*TILE_WIDTH+Col];
	Ps[Row*TILE_WIDTH+Col] = Pvalue;
	__syncthreads();
	
	// Compute on shared memory
	for (int k = 0; k < TILE_WIDTH; k++){
		float M_elem = Ms[Row*TILE_WIDTH + k];
		float N_elem = Ns[k*TILE_WIDTH + Col];
		Ps[Row*TILE_WIDTH + Col] += M_elem * N_elem;
	}
	__syncthreads();

	// Copy back to global memory
	P.elements[Row*TILE_WIDTH + Col] = Ps[Row*TILE_WIDTH + Col];
	//P.elements[0] = TILE_WIDTH;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
